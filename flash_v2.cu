#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/types.h>

__global__ void forward_kernel_v2(const float* Q, const float* K,
                                  const float* V, const int N, const int d,
                                  const int Tc, const int Tr, const int Bc,
                                  const int Br, const float softmax_scale,
                                  float* l, float* m, float* O) {
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int by = blockIdx.y;  // batch and head index

  // Offset into Q,K,V,O,l,m - different for each batch and head
  int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
  int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for l and m

  // Define SRAM for Q,K,V,S
  extern __shared__ float sram[];
  const int tile_size_qo = Br * d;  // size of Qi, Oi
  const int tile_size_kv = Bc * d;  // size of Kj, Vj
  float* Qi = sram;
  float* Oi = &sram[tile_size_qo];
  float* Kj = &sram[tile_size_qo * 2];
  float* Vj = &sram[tile_size_qo * 2 + tile_size_kv];
  float* S = &sram[tile_size_qo * 2 + tile_size_kv * 2];

  for (int i = 0; i < Tr; i++) {
    // Load Kj, Vj to SRAM
    for (int x = 0; x < d; x++) {
      Qi[(tx * d) + x] = Q[qkv_offset + (tile_size_qo * i) + (tx * d) + x];
      Oi[(tx * d) + x] = O[qkv_offset + (tile_size_qo * i) + (tx * d) + x];
    }
    __syncthreads();  // such that the inner loop can use the correct Kj, Vj
    float row_m_prev = m[lm_offset + (Br * i) + tx];
    float row_l_prev = l[lm_offset + (Br * i) + tx];
    float row_m_new, row_l_new;

    for (int j = 0; j < Tc; j++) {
      // Load Qi to SRAM, l and m to registers
      for (int x = 0; x < d; x++) {
        Kj[(tx * d) + x] = K[qkv_offset + (tile_size_kv * j) + (tx * d) + x];
        Vj[(tx * d) + x] = V[qkv_offset + (tile_size_kv * j) + (tx * d) + x];
      }

      // S = QK^T, row_m = rowmax(S)
      float row_m = -INFINITY;
      for (int y = 0; y < Bc; y++) {
        float sum = 0;
        for (int x = 0; x < d; x++) {
          sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
        }
        sum *= softmax_scale;
        S[(Bc * tx) + y] = sum;

        if (sum > row_m) row_m = sum;
      }

      // max mi
      row_m_new = max(row_m_prev, row_m);

      // P = exp(S - row_m), row_l = rowsum(P)
      float row_l = 0;
      for (int y = 0; y < Bc; y++) {
        S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m_new);
        row_l += S[(Bc * tx) + y];
      }

      // Compute new l
      row_l_prev = (__expf(row_m_prev - row_m_new) * row_l_prev) + row_l;

      // Write O, l, m to HBM
      for (int x = 0; x < d; x++) {
        float pv = 0;  // Pij * Vj
        for (int y = 0; y < Bc; y++) {
          pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
        }
        Oi[(tx * d) + x] =
            ((__expf(row_m_prev - row_m_new) * Oi[(tx * d) + x]) + pv);
      }

      row_m_prev = row_m_new;
    }
    for (int x = 0; x < d; x++) {
      O[qkv_offset + (tile_size_qo * i) + (tx * d) + x] =
          Oi[(tx * d) + x] / row_l_prev;
    }
    __syncthreads();
  }
}

torch::Tensor forward_v2(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
  // TODO: determine Bc, Br dynamically
  const int Bc = 32;
  const int Br = 32;

  const int B = Q.size(0);
  const int nh = Q.size(1);
  const int N = Q.size(2);
  const int d = Q.size(3);

  const int Tc = ceil((float)N / Bc);
  const int Tr = ceil((float)N / Br);
  const float softmax_scale = 1.0 / sqrt(d);

  // Initialize O, l, m to HBM
  auto O = torch::zeros_like(Q);
  auto l = torch::zeros({B, nh, N});
  auto m = torch::full({B, nh, N}, -INFINITY);
  torch::Device device(torch::kCUDA);
  l = l.to(device);
  m = m.to(device);

  // Calculate SRAM size needed per block
  const int sram_size = (2 * Br * d * sizeof(float)) +
                        (2 * Bc * d * sizeof(float)) +
                        (Bc * Br * sizeof(float));
  int max_sram_size;
  hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
  printf("Max shared memory: %d, requested shared memory: %d \\n",
         max_sram_size, sram_size);

  dim3 grid_dim(B, nh);  // batch_size x num_heads
  dim3 block_dim(Bc);    // Bc threads per block

  forward_kernel_v2<<<grid_dim, block_dim, sram_size>>>(
      Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(), N, d, Tc,
      Tr, Bc, Br, softmax_scale, l.data_ptr<float>(), m.data_ptr<float>(),
      O.data_ptr<float>());
  return O;
}